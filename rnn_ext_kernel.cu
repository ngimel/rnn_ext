#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include "ATen/cuda/HIPContext.h"
#include "hipblas.h"
#include <ATen/cuda/CUDAEvent.h>
#include <ATen/core/Half.h>

namespace at { namespace native {

    namespace {
    template<typename scalar_t>
    __global__ void add(scalar_t * in1, scalar_t * in2, scalar_t * out, int n){
         int tid = threadIdx.x+blockIdx.x*blockDim.x;
         for (int i=tid; i<n; i += blockDim.x*gridDim.x){
            out[i] = in1[i]+in2[i];
         }
    }
    
    }



    at::Tensor run_gemms(const Tensor& a, const Tensor& h, const Tensor& w1, const Tensor& w2, const bool use_streams){
       int64_t seq_length = a.size(0);
       int64_t batch_size = a.size(1);
       auto blasHandle = at::cuda::getCurrentCUDABlasHandle();
       hipblasSetMathMode(blasHandle, HIPBLAS_TENSOR_OP_MATH);
       // default constructors are now illegal - create something and reassign if using streams
       c10::cuda::CUDAStream s1 = c10::cuda::getDefaultCUDAStream();
       c10::cuda::CUDAStream s2 = c10::cuda::getDefaultCUDAStream();
       at::cuda::CUDAEvent event;
       bool isHighPriority = false;
       int device = 0;
       if (use_streams == true){
           s1 = c10::cuda::getStreamFromPool(isHighPriority, device);
           s2 = c10::cuda::getStreamFromPool(isHighPriority, device);
       }
//gemm: 
       int m = w1.size(1);
       int k = w1.size(0);
       int n = h.size(0);
       Tensor out1 = at::empty_like(h);
       Tensor out2 = at::empty_like(h);
       hipblasStatus_t err;
       for (int t=0; t< seq_length; t++){
           auto in_a = a[t];
           hipStream_t  ss1, ss2=nullptr;
           if (use_streams) {
             ss1  = s1.stream();
             hipblasSetStream(blasHandle, ss1);    
           }
           //submit first gemm
           float alpha = 1.f;
           float beta = 0.f;
           err = hipblasGemmEx(blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                   m, n, k, &alpha,
                                   w1.data<at::Half>(), HIP_R_16F, m, in_a.data<at::Half>(), HIP_R_16F,
                                   k, &beta, out1.data<at::Half>(), HIP_R_16F, m,
                                   HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP);
           AT_CHECK(err==0, "error in cublas"); 
           if (use_streams) {
              event.record(s1);
              ss2 = s2.stream();
              hipblasSetStream(blasHandle, ss2);
           } 
             
           //submit second gemm    
           err = hipblasGemmEx(blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                   m, n, k, &alpha,
                                   w2.data<at::Half>(), HIP_R_16F, m, h.data<at::Half>(), HIP_R_16F,
                                   k, &beta, out2.data<at::Half>(), HIP_R_16F, m,
                                   HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP);
           AT_CHECK(err==0, "error in cublas"); 
           if (use_streams) {
              event.block(s2);
           }
        
          int numThreads = 256;
          int numBlocks = (h.numel()+numThreads -1)/numThreads;
          AT_DISPATCH_ALL_TYPES_AND_HALF(a.type(), "add", [&] {
           add<<<numBlocks, numThreads, 0, ss2>>>(
           out1.data<scalar_t>(), out2.data<scalar_t>(), h.data<scalar_t>() , h.numel());
 
          });
       }
      
       
       
       return h;
    }
}}
